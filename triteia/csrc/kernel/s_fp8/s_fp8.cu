#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"
#include "cutlass/arch/mma_sm90.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
